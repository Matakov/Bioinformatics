#include "hip/hip_runtime.h"
#include"SW.h"
#include"utility.h"

// CUDA kernel to add elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride) y[i] = x[i] + y[i];
}


extern "C" void calculate(float *x,float *y, int N)
{
  // Allocate Unified Memory -- accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));
 
  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }
 
  // Launch kernel on 1M elements on the GPU
  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;
  add<<<numBlocks, blockSize>>>(N, x, y);
 
  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
 
  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;
 
  // Free memory
  hipFree(x);
  hipFree(y);
 
  return;
}

/*
Authors: Franjo Matkovic

Parameters:
    input: c++ string
    output: char array
-Function to allocate unified memory
 and copy to unified memory
*/
char* allocateMemory(std::string const& x)
{
    char* memory;
    const char *cstr = x.c_str();
    hipMallocManaged(&memory, x.length()*(sizeof(char)+1));
    //x.copy( memory, x.length() );
    //for(int i=0;i<x.length();++i) memory[i]=cstr[i];
    strcpy(memory, cstr);   
    memory[x.length()]='\0';
    //for(int i=0;i<x.length();++i) std::cout<<cstr[i];
    //std::cout<<"Memory allocated"<<std::endl;
    //for(int i=0;i<x.length();++i) std::cout<<memory[i];
    //std::cout<<std::endl;
    return memory;
}


/*
Authors: Franjo Matkovic

Parameters:
    input: c++ string
    output: char array
-Function to allocate cuda memory
*/
extern "C" float* allocateMatrixMemory(const std::string& x,const std::string& y)
{
    float* memory;
    //hipMallocManaged(&memory, (x.length()+1)*(y.length()+1)*(sizeof(float)));
    //x.copy( memory, x.length() );
    //for(int i=0;i<x.length();++i) memory[i]=cstr[i];
    hipMalloc((float **)&memory, (x.length()+1) * (y.length()+1) * sizeof(float)); 
    //for(int i=0;i<x.length();++i) std::cout<<cstr[i];
    //std::cout<<"Memory allocated"<<std::endl;
    //for(int i=0;i<x.length();++i) std::cout<<memory[i];
    //std::cout<<std::endl;
    return memory;
}

/*
Authors: Franjo Matkovic

Parameters:
    input: c++ string
    output: char array
-Function to allocate memory
*/
extern "C" float* allocateMatrixMemoryCPU(const std::string& x,const std::string& y)
{
    float* memory;
    //hipMallocManaged(&memory, (x.length()+1)*(y.length()+1)*(sizeof(float)));
    //x.copy( memory, x.length() );
    //for(int i=0;i<x.length();++i) memory[i]=cstr[i];
    memory =(float *) malloc((x.length()+1) * (y.length()+1) * sizeof(float));  
    //for(int i=0;i<x.length();++i) std::cout<<cstr[i];
    //std::cout<<"Memory allocated"<<std::endl;
    //for(int i=0;i<x.length();++i) std::cout<<memory[i];
    //std::cout<<std::endl;
    return memory;
}

/*
Authors: Franjo Matkovic

Parameters:
*/
extern "C" float* initializeMemoryMatrixCPU(const std::string& x,const std::string& y, double penalty)
{
    double d=penalty;
    double e=penalty;
    float* memory = allocateMatrixMemoryCPU( x, y);
    for(int i=0;i<x.length()+1;i++)
    {
        for(int j=0;j<y.length()+1;j++)
        {
            //printf("%d,%d\n",i,j);
            if(i==0)
            {
                memory[i*(x.length()+1)+j] = -(d+e*(j-1));
            }
            else if(j==0)
            {   
                memory[i*(x.length()+1)+j] = -(d+e*(i-1));
            }
            else
            {
                memory[i*(x.length()+1)+j] = 0;
            }
            //printf("%d,%d: %f\n",i,j,memory[i*x.length()+j]);
        }
    }
    memory[0] = 0;
    return memory;
}

/*
Authors: Franjo Matkovic

Parameters:
    input: array pointer
    output: -
-Function to release unified memory
*/
extern "C" void releaseMemory(char* memory)
{
    hipFree(memory);
    //std::cout<<"Memory released"<<std::endl;
    return;
}

__device__ volatile int sem = 0;

__device__ void acquire_semaphore(volatile int *lock)
{
    while (atomicCAS((int *)lock, 0, 1) != 0);
}

__device__ void release_semaphore(volatile int *lock)
{
    *lock = 0;
    __threadfence();
}

__global__ void initmemoryHNW(double *memory,long int const m,long int const n, double const d, double const e, double const N)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    //printf("Hello from block %d, thread %d, index %d, Memory is %f\n", blockIdx.x, threadIdx.x,index,memory[index]);
   
    for (int i = index; i < N; i += stride)
    {
        if(i<n)
        {
            memory[i]=-(d+e*(i-1));
        }
        else if(i%n==0)
        {
            memory[i]=-(d+e*(i/n-1));
        }
        else
        {
            memory[i]=0;
        }
    }
    return;
}
/*
__global__ void initmemoryHSW(double *memory,long int const m,long int const n, double const d, double const e, double const N)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    //printf("Hello from block %d, thread %d, index %d, Memory is %f\n", blockIdx.x, threadIdx.x,index,memory[index]);
   
    for (int i = index; i < N; i += stride)
    {
        memory[i]=0;
    }
    return;
}
*/
/*
Authors: Matej Crnac

Parameters:
    input:  semaphor - pointer to semaphor list
            n - seamphor length
    output: - initialised semaphor.
*/
__global__ void initsemaphor(int *semaphore, int N)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    //printf("Hello from block %d, thread %d, index %d, Memory is %f\n", blockIdx.x, threadIdx.x,index,memory[index]);
   
    for (int i = index; i < N; i += stride)
    {
            semaphore[i]=0;
    }
    return;
}

__global__ void initmemoryHSW(double *memory,double const m,double const n, double const d, double const e, double const N)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += stride)
    {
        memory[i]=0;
    }
    return;
}

/*
Authors: Matej Crnac

Parameters:
    input:  index - matrix index
            n - s2 length + 1
    output: - index of character in left string for given matrix index.
*/
__device__ long int find_index_left(long int index,long int n)
{
    return index/n - 1;
}

/*
Authors: Matej Crnac

Parameters:
    input:  index - matrix index
            n - s2 length + 1
    output: - index of character in upper string for given matrix index.
*/
__device__ long int find_index_upper(long int index,long int n)
{
    return index%n - 1;
}

/*
Authors: Matej Crnac, Franjo Matković

Parameters:
    input:  memory - pointer to matrix
            m - s1 length + 1
            n - s2 length + 1
            d - penalty
            e - penalty
            N - matrix size
            sim - similarity function
            s1 - string 1
            s2 - string 2
    output: - solved cost matrix
-Function to solve NeedlemanWunsch using GPU
*/
__global__ void NW_GPU(double* memory,long int const m,long int const n, double const d, double const e, long int const N,double (*sim)(char,char),const char* s1, const char* s2,int* semaphore)
{
    //extern __shared__ int s[];
    printf("Bez: %d, blockDim: %d, n = %ld\n",(int)n,blockDim.x,n);
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    printf("index: %d Stride: %d N: %ld\n",index,stride,N);
    //printf("%f\n",(float)(m-1)/(blockDim.x/2));
    //printf("blockIdx.x-int((double)(n-1)/(blockDim.x/2)) = %f\n",blockIdx.x-int((double)(n-1)/(blockDim.x/2)));
    //printf("semaphore[blockIdx.x-int((double)(n-1)/(blockDim.x/2))] = %f \n",semaphore[blockIdx.x-int((double)(n-1)/(blockDim.x/2))]);
    //printf("index = %d, blockidx = %d, Semaphor = %d\n",index,blockIdx.x,semaphore[blockIdx.x]);
    while(1)
    {
        __syncthreads();
        if(blockIdx.x==0) break;
        //if(blockIdx.x<(int)(double)(n-1)/(blockDim.x/2) && semaphore[blockIdx.x-1]==1) break;
        //if(blockIdx.x>=(int)(double)(n-1)/(blockDim.x/2) && semaphore[blockIdx.x-1]==1 && semaphore[blockIdx.x-int((double)(n-1)/(blockDim.x/2))]==1) break;
        //if(blockIdx.x%(int)(double)(n-1)/(blockDim.x/2)==0 && semaphore[blockIdx.x-int((double)(n-1)/(blockDim.x/2))]==1) break;

        if(blockIdx.x<(int)n && semaphore[blockIdx.x-1]==1) break;
        if(blockIdx.x>=(int)n && semaphore[blockIdx.x-1]==1 && semaphore[blockIdx.x-int(n)]==1) break;
        if(blockIdx.x%(int)n==0 && semaphore[blockIdx.x-int(n)]==1) break;
    }
    for (int i = index; i < N; i += stride)
    {
        if(i%n!=0 && i > n)
        {   
            double simil;
            if(s1[find_index_left(i,n)]==s2[find_index_upper(i,n)]) simil = 1;
            else simil = -3;
            //printf("Hello from block %d, thread %d, index %d, Memory is %f\n", blockIdx.x, threadIdx.x,index,memory[index]);
            //printf("Index: %d\n", i);
            //printf("memory[i-n-1] = %f\n", memory[i-n-1]);
            //printf("find_L:\n");
            //printf("find_ind_l = %d\n",find_index_left(i,n));
            //printf("s1[find_index_left(i,n)] = %c\n",s1[find_index_left(i,n)]);
            //printf("s1 finished\n");
            //printf("s2[find_index_upper(i,n)] = %c\n",s2[find_index_upper(i,n)]);
            //printf("s2 finished\n");
            //printf("sim: = %f\n",simil);
            //printf("sim finished\n");
            //printf("Index: %d, memory[i-n-1] = %f, sim: %d find_ind_l = %d, find_ind_u = %d, memory[i-n] = %f, memory[i-1] = %f\n",i, memory[i-n-1],simil, find_index_left(i,n), find_index_upper(i,n), memory[i-n], memory[i-1]);
            memory[i]=max(memory[i-n-1]+simil,max(memory[i-n] - d,memory[i-1] - d));
        }
    }
    
    semaphore[blockIdx.x]=1;
    //semaphore[blockIdx.x+1]=1;
    //semaphore[blockIdx.x+n]=1;
    

    return;
}


/*
Authors: Franjo Matkovic

Parameters:
    input:  s1 - string 1
            s2 - string 2
            d  - penalty
            e  - penalty
            sim- similarity function
    output: - solved cost matrix
-Function to solve NeedlemanWunsch
*/
void NeedlemanWunschGPU(std::string const& s1, std::string const& s2, double const d, double const e,double (*sim)(char,char))
{
    double *Gi,*Gd,*F,*E;
    double *memory;
    char *M;
    long int m = s1.length();
    long int n = s2.length();
    long int N = (s1.length()+1)*(s2.length()+1);
    //long int N_orig = n*m;
    hipMallocManaged(&memory, N*sizeof(double));
    hipMallocManaged(&M, N*sizeof(char));
    hipMallocManaged(&Gi, N*sizeof(double));
    hipMallocManaged(&Gd, N*sizeof(double));
    hipMallocManaged(&F, N*sizeof(double));
    hipMallocManaged(&E, N*sizeof(double));
    
    int blockSize = 1;
    int numBlocks = (N + blockSize - 1) / blockSize;
    std::cout<<numBlocks<<std::endl;


    int *semaphore;

    hipMallocManaged(&semaphore, numBlocks);

    initsemaphor<<<numBlocks, blockSize>>>(semaphore, numBlocks);
    hipDeviceSynchronize();

    initmemoryHNW<<<numBlocks, blockSize>>>(memory,m+1,n+1,d,e,N);
    hipDeviceSynchronize();
    
    //padding(s1,s2,,);
    const char* x1 = allocateMemory(s1);
    const char* x2 = allocateMemory(s2);
    //int i = 0;

    /*while( x2[i] != '\0')
    {
        std::cout<<x1[i];
        i++;
    }*/
    
    std::cout<<"Seamphor before:"<<" ";
    for(int i=0;i<numBlocks;i++)
    {
        std::cout<<semaphore[i]<<" ";    
    }
    std::cout<<std::endl;
 
    NW_GPU<<<numBlocks, blockSize>>>(memory,m+1,n+1,d,e,N,sim,x1,x2,semaphore); 
    hipDeviceSynchronize();
    
    std::cout<<"Seamphor after:"<<" ";
    for(int i=0;i<numBlocks;i++)
    {
        std::cout<<semaphore[i]<<" ";    
    }
    std::cout<<std::endl;

    for(int i=0;i<m+1;i++)
    {
        for(int j=0;j<n+1;j++)
        {
            std::cout<<memory[i*(n+1)+j]<<" ";  
        }
        std::cout<<std::endl;   
    }
    //memory freeing
    hipFree(memory);
    hipFree(M);
    hipFree(Gi);
    hipFree(Gd);
    hipFree(F);
    hipFree(E);
    return;
} 

/*
Authors: Matej Crnac, Franjo Matković

Parameters:
    input:  memory - pointer to matrix
            m - s1 length + 1
            n - s2 length + 1
            d - penalty
            e - penalty
            N - matrix size
            sim - similarity function
            s1 - string 1
            s2 - string 2
    output: - solved cost matrix
-Function to solve SmithWaterman using GPU
*/
__global__ void SW_GPU(double* memory,long int const m,long int const n, double const d, double const e, long int const N,const char* s1, const char* s2,int* semaphore)
{
    //extern __shared__ int s[];
    //printf("Bez: %d, blockDim: %d, n = %ld\n",(int)n,blockDim.x,n);
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    //printf("index: %d Stride: %d N: %ld\n",index,stride,N);
    //printf("%f\n",(float)(m-1)/(blockDim.x/2));
    //printf("blockIdx.x-int((double)(n-1)/(blockDim.x/2)) = %f\n",blockIdx.x-int((double)(n-1)/(blockDim.x/2)));
    //printf("semaphore[blockIdx.x-int((double)(n-1)/(blockDim.x/2))] = %f \n",semaphore[blockIdx.x-int((double)(n-1)/(blockDim.x/2))]);
    //printf("index = %d, blockidx = %d, Semaphor = %d\n",index,blockIdx.x,semaphore[blockIdx.x]);
    //printf("gridDim: %d\n",gridDim.x);
    for (int i = index; i < N; i += stride)
    {
        __syncthreads();
        while(1)
        {
            
            //printf("index = %d i = %d, blockidx = %d, threadID = %d, Semaphor = %d, memory = %f\n",index,i,blockIdx.x,threadIdx.x,semaphore[blockIdx.x],memory[i-1]);
            if(i==0) break;
            //if(blockIdx.x<(int)(double)(n-1)/(blockDim.x/2) && semaphore[blockIdx.x-1]==1) break;
            //if(blockIdx.x>=(int)(double)(n-1)/(blockDim.x/2) && semaphore[blockIdx.x-1]==1 && semaphore[blockIdx.x-int((double)(n-1)/(blockDim.x/2))]==1) break;
            //if(blockIdx.x%(int)(double)(n-1)/(blockDim.x/2)==0 && semaphore[blockIdx.x-int((double)(n-1)/(blockDim.x/2))]==1) break;

            if(i<(int)n && semaphore[(i-1)%gridDim.x]>0)
            {
                semaphore[(i-1)%gridDim.x]--;
                break;
            }
            if(i>=(int)n && semaphore[(i-1)%gridDim.x]>0 && semaphore[(i-int(n))%gridDim.x]>0)
            {
                semaphore[(i-int(n))%gridDim.x]--;
                semaphore[(i-1)%gridDim.x]--;
                break;
            }
            if(i%(int)n==0 && semaphore[(i-int(n))%gridDim.x]>0)
            {
                semaphore[(i-int(n))%gridDim.x]--;
                break;
            }
        }
        if(i%n!=0 && i > n)
        {   
            double simil;
            if(s1[find_index_left(i,n)]==s2[find_index_upper(i,n)]) simil = 1;
            else simil = -3;
            //printf("Hello from block %d, thread %d, index %d, Memory is %f\n", blockIdx.x, threadIdx.x,index,memory[index]);
            //printf("Index: %d\n", i);
            //printf("memory[i-n-1] = %f\n", memory[i-n-1]);
            //printf("find_L:\n");
            //printf("find_ind_l = %d\n",find_index_left(i,n));
            //printf("s1[find_index_left(i,n)] = %c\n",s1[find_index_left(i,n)]);
            //printf("s1 finished\n");
            //printf("s2[find_index_upper(i,n)] = %c\n",s2[find_index_upper(i,n)]);
            //printf("s2 finished\n");
            //printf("sim: = %f\n",simil);
            //printf("sim finished\n");
            //printf("Index: %d, memory[i-n-1] = %f, sim: %d find_ind_l = %d, find_ind_u = %d, memory[i-n] = %f, memory[i-1] = %f\n",i, memory[i-n-1],simil, find_index_left(i,n), find_index_upper(i,n), memory[i-n], memory[i-1]);
            memory[i]=max((double)0,max(memory[i-n-1]+simil,max(memory[i-n] - d,memory[i-1] - d)));
        }
        semaphore[(blockIdx.x)%gridDim.x]=2;
    }
    
    
    //semaphore[blockIdx.x+1]=1;
    //semaphore[blockIdx.x+n]=1;
    

    return;
}

/*
Authors: Franjo Matkovic

Parameters:
    input:  s1 - string 1
            s2 - string 2
            d  - penalty
            e  - penalty
            sim- similarity function
    output: - solved cost matrix
-Function to solve SmithWaterman
*/
/*
void SmithWatermanGPU(std::string const& s1, std::string const& s2, double const d, double const e)
{
    double *Gi,*Gd,*F,*E;
    double *memory;
    char *M;
    long int m = s1.length();
    long int n = s2.length();
    long int N = (s1.length()+1)*(s2.length()+1);
    long int N_orig = n*m;
    hipMallocManaged(&memory, N*sizeof(double));
    hipMallocManaged(&M, N*sizeof(char));
    hipMallocManaged(&Gi, N*sizeof(double));
    hipMallocManaged(&Gd, N*sizeof(double));
    hipMallocManaged(&F, N*sizeof(double));
    hipMallocManaged(&E, N*sizeof(double));
    
    int blockSize = 1;
    int numBlocks;
    if (N <= 240) {
        numBlocks = N;
    }
    else {
        numBlocks = 240;
    }
    std::cout<<numBlocks<<std::endl;


    int *semaphore;

    hipMallocManaged(&semaphore, numBlocks);

    initsemaphor<<<numBlocks, blockSize>>>(semaphore, numBlocks);
    hipDeviceSynchronize();

    initmemoryHSW<<<numBlocks, blockSize>>>(memory,m+1,n+1,d,e,N);
    hipDeviceSynchronize();
    
    //padding(s1,s2,,);
    const char* x1 = allocateMemory(s1);
    const char* x2 = allocateMemory(s2);
    //int i = 0;

    while( x2[i] != '\0')
    {
        std::cout<<x1[i];
        i++;
    }
    
    std::cout<<"Seamphore before:"<<" ";
    for(int i=0;i<numBlocks;i++)
    {
        std::cout<<semaphore[i]<<" ";    
    }
    std::cout<<std::endl;
    
    SW_GPU<<<numBlocks, blockSize>>>(memory,m+1,n+1,d,e,N,x1,x2,semaphore); 
    hipDeviceSynchronize();
    
    std::cout<<"Seamphore after:"<<" ";
    for(int i=0;i<numBlocks;i++)
    {
        std::cout<<semaphore[i]<<" ";    
    }
    std::cout<<std::endl;
    
    for(int i=0;i<m+1;i++)
    {
        for(int j=0;j<n+1;j++)
        {
            std::cout<<memory[i*(n+1)+j]<<" ";  
        }
        std::cout<<std::endl;   
    }
    //memory freeing
    hipFree(memory);
    hipFree(M);
    hipFree(Gi);
    hipFree(Gd);
    hipFree(F);
    hipFree(E);
    return;
} 
*/

/*
Authors: Dario Sitnik, Franjo Matković

Parameters:
    input:  memory - pointer to matrix
            m - s1 length + 1
            n - s2 length + 1
            d - penalty
            e - penalty
            N - matrix size
            sim - similarity function
            s1 - string 1
            s2 - string 2
    output: - solved cost matrix
-Function to solve SmithWaterman using GPU on thread level
*/

__global__ void threadSolver(float *memory,long int subM,long int subN, long int const n, float const d, float const e, long int const b_size, char *s1, char *s2, int *semaphore)
{
	long int index = (threadIdx.x/b_size + subM) * n + subN + threadIdx.x % b_size;
	long int last = (subM + b_size - 1) * n + subN + b_size - 1;

	/*
	printf("%ld, %ld",index,last);
	for(long int i=index;i<last;i += blockDim.x)
	{
	__syncthreads();
	while(1)
	{
	    
	    //printf("index = %d i = %d, blockidx = %d, threadID = %d, Semaphor = %d, memory = %f\n",index,i,blockIdx.x,threadIdx.x,semaphore[blockIdx.x],memory[i-1]);
	    if(i==0) break;
	    //if(blockIdx.x<(int)(double)(n-1)/(blockDim.x/2) && semaphore[blockIdx.x-1]==1) break;
	    //if(blockIdx.x>=(int)(double)(n-1)/(blockDim.x/2) && semaphore[blockIdx.x-1]==1 && semaphore[blockIdx.x-int((double)(n-1)/(blockDim.x/2))]==1) break;
	    //if(blockIdx.x%(int)(double)(n-1)/(blockDim.x/2)==0 && semaphore[blockIdx.x-int((double)(n-1)/(blockDim.x/2))]==1) break;

	    if(i<(int)n && semaphore[(i-1)%blockDim.x]>0)
	    {
		semaphore[(i-1)%blockDim.x]--;
		break;
	    }
	    if(i>=(int)n && semaphore[(i-1)%blockDim.x]>0 && semaphore[(i-int(n))%blockDim.x]>0)
	    {
		semaphore[(i-int(n))%blockDim.x]--;
		semaphore[(i-1)%blockDim.x]--;
		break;
	    }
	    if(i%(int)n==0 && semaphore[(i-int(n))%blockDim.x]>0)
	    {
		semaphore[(i-int(n))%blockDim.x]--;
		break;
	    }
	}
	if(i%n!=0 && i > n)
	{   
	    float simil;
	    if(s1[find_index_left(i,n)]==s2[find_index_upper(i,n)]) simil = 1;
	    else simil = -3;
	    memory[i]=max((float)0,max(memory[i-n-1]+simil,max(memory[i-n] - d,memory[i-1] - d)));
	}
	semaphore[(threadIdx.x)%blockDim.x]=2;
	}
	*/
	return;
}

//block 
__global__ void kernelCallsKernel(float *memory,long int const m,long int const n, float const d, float const e, long int N, char *s1, char *s2, int numBlocks)
{
    //b_size = n / sqrt(numBlocks);
    long int subM = blockIdx.x;
    long int subN;
    int *semaphore;
    //initsemaphor(semaphore,blockDim.x);
    //blockIdx.x //određuje blok
    //threadSolver<<<1,4>>>(memory,subM,subN,n,d,e,b_size,s1,s2,semaphore);
    return;
} 

void SmithWatermanGPU(std::string const& s1, std::string const& s2, double const d, double const e, double const B)
{
	//input strings are const so we copy
	std::string string_m(s1);
	std::string string_n(s2);

	//memory locations 
	float *Gi,*Gd,*F,*E;
	float *memory;
	char *M;

	//sizes of strings
	long int m = string_m.length();
	long int n = string_n.length();


	//B is the desirable number of blocks in grid
	double k = sqrt(B/(m/n));
	long int blockSize_n = floor(k);
	long int blockSize_m = floor((m/n)*k);
	long int blockSize = blockSize_n*blockSize_m;

	//std::cout<<k<<" "<<blockSize_n<<" "<<blockSize_m<<std::endl;
	//here we define how much will there be blocks in m and n direction
	long int blockNum_n = ceil((double)n/blockSize_n);
	long int blockNum_m = ceil((double)m/blockSize_m);	
	long int blockNum = blockNum_m*blockNum_n;

	//std::cout<<"Size:"<<n<<" "<<blockSize_n<<" "<<ceil((double)n/blockSize_n)<<" "<<ceil(n/blockSize_n)<<std::endl;
	//std::cout<<"Size:"<<m<<" "<<blockSize_m<<" "<<ceil((double)m/blockSize_m)<<" "<<ceil(m/blockSize_m)<<std::endl;
	//here we are padding strings so there are no elements that will be
 		 
	padding(string_m,string_n,blockNum_m*blockSize_m,blockNum_n*blockSize_n);
	//std::cout<<string_m<<std::endl;
	//std::cout<<string_n<<std::endl;
	//std::cout<<"Size:"<<string_m.length()<<" "<<string_n.length()<<std::endl;
	
	//strings have been padded so their length is measured again	
	m=string_m.length();
	n=string_n.length();	

	long int N = (m+1)*(n+1);
	//part of code where memory allocation is happening
	hipMallocManaged(&memory, N*sizeof(float));
	hipMallocManaged(&M, N*sizeof(char));
	hipMallocManaged(&Gi, N*sizeof(float));
	hipMallocManaged(&Gd, N*sizeof(float));
	hipMallocManaged(&F, N*sizeof(float));
	hipMallocManaged(&E, N*sizeof(float));
	
	char* x1 ;//= allocateMemory(string_m);
	
	const char *cstr = string_m.c_str();
    	hipMallocManaged(&x1, string_m.length()*(sizeof(char)+1));
    	//x.copy( memory, x.length() );
    	//for(int i=0;i<x.length();++i) memory[i]=cstr[i];
    	strcpy(x1, cstr);   
    	x1[string_m.length()]='\0';
	
    	char* x2 ;// = allocateMemory(string_n);
	
	const char *cstr2 = string_n.c_str();
	hipMallocManaged(&x2, string_n.length()*(sizeof(char)+1));
    	//x.copy( memory, x.length() );
    	//for(int i=0;i<x.length();++i) memory[i]=cstr[i];
    	strcpy(x2, cstr2);   
    	x2[string_n.length()]='\0';
	
	int *semaphore;
	blockSize = 64;
	std::cout<<blockNum<<" "<<blockSize<<std::endl;
	initsemaphor<<<1, 64>>>(semaphore, blockSize);
    	hipDeviceSynchronize();
	//blockSize_m,blockSize_n,blockNum_m,blockNum_n
	threadSolver<<<1, 64>>>(memory,0,0,n,d,e,5,x1,x2,semaphore);
	
	//threadSolver(float *memory,long int subM,long int subN, long int const n, float const d, float const e, long int const b_size, char *s1, char *s2, int *semaphore)
	for(int i=0;i<N;i++)
	{
		std::cout<<memory[i]<<" ";
	}

	//memory freeing
	hipFree(memory);
	hipFree(M);
	hipFree(Gi);
	hipFree(Gd);
	hipFree(F);
	hipFree(E);
	
	return;
} 

/*
void SmithWatermanGPU(std::string const& s1, std::string const& s2, double const d, double const e)
{
	//input strings are const so we copy
	std::string string_m(s1);
	std::string string_n(s2);

	//memory locations 
	double *Gi,*Gd,*F,*E;
	double *memory;
	char *M;

	//this part here defines how much memory elements will one block solve
	long int blockSize = 1024;          // one block will solve 1024 elements
	long int blockSize_m = sqrt(blockSize);     // here we define block size in m direction 
	long int blockSize_n = sqrt(blockSize);     // here we define block size in n direction

	padding(string_m,string_n,m+m%blockSize_m,n+n%blockSize_n);

	//defining sizes of strings that will be compared
	long int m = string_m.length();
	long int n = string_n.length();

	//cost matrix that will be allocated needs to have an extra row and column
	long int N = (string_m.length()+1)*(string_n.length()+1);

	//here we calculate how much blocks will be in each direction
	long int numBlocks_m = m/blockSize_m;       // 
	long int numBlocks_n = n/blockSize_n;       //

	long int numBlocks = numBlocks_m*numBlocks_n;

	//part of code where memory allocation is happening
	hipMallocManaged(&memory, N*sizeof(double));
	hipMallocManaged(&M, N*sizeof(char));
	hipMallocManaged(&Gi, N*sizeof(double));
	hipMallocManaged(&Gd, N*sizeof(double));
	hipMallocManaged(&F, N*sizeof(double));
	hipMallocManaged(&E, N*sizeof(double));

	//now we have block sizes in m and n direction along with number of blocks in m and n direction




	//memory freeing
	hipFree(memory);
	hipFree(M);
	hipFree(Gi);
	hipFree(Gd);
	hipFree(F);
	hipFree(E);
	return;
}
*/
